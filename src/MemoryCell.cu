#include "MemoryCell.cuh"

long long MemoryCell::n = 0;

MemoryCell::MemoryCell(int c) {
	nConnections = c;
	activationIn = 0; activationInPrime = 0;
	activationOut = 0; activationOutPrime = 0;
	state = 0; previousState = 0;
	feedback = 0; previousFeedback = 0;
	bias = 0;

	default_random_engine g(time(0) + (n++));
	normal_distribution<double> d(0, 1);

	cell_hidden_weight = d(g);
	cell_hidden_partial = 0;
	input_hidden_partial = 0;
	forget_hidden_partial = 0;

	cell_data_weight = (double *)malloc(sizeof(double) * c);
	cell_data_partial = (double *)malloc(sizeof(double) * c);
	forget_data_partial = (double *)malloc(sizeof(double) * c);
	input_data_partial = (double *)malloc(sizeof(double) * c);

	for (int i = 0; i < c; i++) {
		cell_data_weight[i] = (d(g));
		cell_data_partial[i] = (0);
		forget_data_partial[i] = (0);
		input_data_partial[i] = (0);
	}
}

MemoryCell::~MemoryCell() {}

__device__ double MemoryCell::activateIn(double data) {
	activationIn = activationFunction(data);
	activationInPrime = activationFunctionPrime(data);
	return activationIn;
}

__device__ double MemoryCell::activateOut(double data) {
	activationOut = activationFunction(data);
	activationOutPrime = activationFunctionPrime(data);
	return activationOut;
}

MemoryCell *MemoryCell::copyToGPU(MemoryCell *memory) {
	MemoryCell *memoryCell;
	hipMalloc((void **)&memoryCell, (sizeof(MemoryCell)));
	hipDeviceSynchronize();
	hipMemcpy(memoryCell, memory, sizeof(MemoryCell), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	double *cdw, *idp, *fdp, *cdp;
	hipMalloc((void **)&cdw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&idp, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&fdp, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&cdp, (sizeof(double) * memory->nConnections));
	hipDeviceSynchronize();

	hipMemcpy(cdw, memory->cell_data_weight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(idp, memory->input_data_partial, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(fdp, memory->forget_data_partial, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(cdp, memory->cell_data_partial, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipMemcpy(&(memoryCell->cell_data_weight), &cdw, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryCell->input_data_partial), &idp, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryCell->forget_data_partial), &fdp, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryCell->cell_data_partial), &cdp, sizeof(double *), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	return memoryCell;
}

MemoryCell *MemoryCell::copyFromGPU(MemoryCell *memory) {

	MemoryCell *memoryCell;
	memoryCell = (MemoryCell *)malloc((sizeof(MemoryCell)));
	hipDeviceSynchronize();
	hipMemcpy(memoryCell, memory, sizeof(MemoryCell), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	double *cdw, *idp, *fdp, *cdp;
	cdw = (double *)malloc(sizeof(double) * memoryCell->nConnections);
	idp = (double *)malloc(sizeof(double) * memoryCell->nConnections);
	fdp = (double *)malloc(sizeof(double) * memoryCell->nConnections);
	cdp = (double *)malloc(sizeof(double) * memoryCell->nConnections);

	hipMemcpy(cdw, memoryCell->cell_data_weight, (sizeof(double) * memoryCell->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(idp, memoryCell->input_data_partial, (sizeof(double) * memoryCell->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(fdp, memoryCell->forget_data_partial, (sizeof(double) * memoryCell->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(cdp, memoryCell->cell_data_partial, (sizeof(double) * memoryCell->nConnections), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	memcpy(&(memoryCell->cell_data_weight), &cdw, (sizeof(double *)));
	memcpy(&(memoryCell->input_data_partial), &idp, (sizeof(double *)));
	memcpy(&(memoryCell->forget_data_partial), &fdp, (sizeof(double *)));
	memcpy(&(memoryCell->cell_data_partial), &cdp, (sizeof(double *)));

	return memoryCell;
}

