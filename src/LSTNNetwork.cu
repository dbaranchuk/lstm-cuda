#include "hip/hip_runtime.h"
#include "LSTMNetwork.cuh"

__global__ void forwardPass(Neuron **neurons, double *connections, double *activations, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			activations[idx] = neurons[idx]->forward(connections);
		}
	}
}

__global__ void backwardPass(Neuron **neurons, double *weightedError, double *errorSum, double learningRate, int connections, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			double *contribution = neurons[idx]->backward(weightedError[idx], learningRate);
			for (int j = 0; j < connections; j++) {
				errorSum[j] += contribution[j];
			}
		}
	}
}

__global__ void forwardPassLSTM(MemoryBlock **blocks, double *connections, double *activations, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			double *blockActivation = blocks[idx]->forward(connections);
			for (int j = 0; j < blocks[i]->nCells; j++) activations[idx * blocks[i]->nCells + j] = blockActivation[j];
		}
	}
}

__global__ void backwardPassLSTM(MemoryBlock **blocks, double **weightedError, double *errorSum, double learningRate, int connections, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			double *contribution = blocks[idx]->backward(weightedError[idx], learningRate);
			for (int j = 0; j < connections; j++) {
				errorSum[j] += contribution[j];
			}
		}
	}
}

LSTMNetwork::LSTMNetwork(int is, int b, int c, double lr, int num_classes) {
	inputSize = is;
	learningRate = lr;
	for (int i = 0; i < b; i++) {
		blocks.push_back(MemoryBlock(c, is));
	}
	for (int i = 0; i < num_classes; i++)
		layer.push_back(Neuron(b * c));
}

LSTMNetwork::~LSTMNetwork() {}

vector<double> LSTMNetwork::classify(vector<double> input) {
	double *output = (double *)malloc(sizeof(double) * blocks.size() * blocks[0].nCells),
			*connections;
	hipMalloc((void **)&connections, sizeof(double) * input.size());
	hipMemcpy(&connections[0], &input[0], (sizeof(double) * input.size()), hipMemcpyHostToDevice);
	if (input.size() == inputSize) {
		// calculate activations from bottom up
		double *activations;
		hipMalloc((void **)&activations, (sizeof(double) * blocks.size() * blocks[0].nCells));

		MemoryBlock **deviceBlocks, **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
		for (int i = 0; i < blocks.size(); i++) {
			hipMemcpy(&(blocks[i].impulse[0]), &connections[0], (sizeof(double) * blocks[i].nConnections), hipMemcpyDeviceToHost);
		}
		hipMalloc((void **)&deviceBlocks, sizeof(MemoryBlock *) * blocks.size());
		for (int i = 0; i < blocks.size(); i++) {
			MemoryBlock *db = MemoryBlock::copyToGPU(&blocks[i]);
			hipMemcpy(&deviceBlocks[i], &db, sizeof(MemoryBlock *), hipMemcpyHostToDevice);
		} forwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, connections, activations, blocks.size(),
		                                             ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
		hipDeviceSynchronize();

		hipMemcpy(&blockBuffer[0], &deviceBlocks[0], (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);
		for (int i = 0; i < blocks.size(); i++) {
			blocks[i] = *MemoryBlock::copyFromGPU(blockBuffer[i]);
		} free(blockBuffer);
		hipFree(deviceBlocks);

		hipFree(connections);
		hipMalloc((void **)&connections, (sizeof(double) * blocks.size() * blocks[0].nCells));
		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * blocks.size() * blocks[0].nCells), hipMemcpyDeviceToDevice);
		hipFree(activations);
		free(output);
		output = (double *)malloc(sizeof(double) * layer.size());

        // Layer
		hipMalloc((void **)&activations, (sizeof(double) * layer.size()));

		Neuron **deviceNeurons, **neuronBuffer = (Neuron **)malloc(sizeof(Neuron *) * layer.size());
		for (int j = 0; j < layer.size(); j++) {
			hipMemcpy(&(layer[j].impulse[0]), &connections[0], (sizeof(double) * layer[j].connections), hipMemcpyDeviceToHost);
		}
		hipMalloc((void **)&deviceNeurons, sizeof(Neuron *) * layer.size());
		for (int j = 0; j < layer.size(); j++) {
			Neuron *dn = Neuron::copyToGPU(&layer[j]);
			hipMemcpy(&deviceNeurons[j], &dn, sizeof(Neuron *), hipMemcpyHostToDevice);
		} forwardPass<<<maxBlocks, maxThreads>>>(deviceNeurons, connections, activations, layer.size(), ceil((double)layer.size() / (double)(maxBlocks * maxThreads)));
		hipDeviceSynchronize();

		hipFree(connections);
		hipMalloc((void **)&connections, (sizeof(double) * layer.size()));
		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * layer.size()), hipMemcpyDeviceToDevice);
		hipMemcpy(&neuronBuffer[0], &deviceNeurons[0], (sizeof(Neuron *) * layer.size()), hipMemcpyDeviceToHost);
		for (int j = 0; j < layer.size(); j++) {
			layer[j] = *Neuron::copyFromGPU(neuronBuffer[j]);
		} 
		hipMemcpy(&output[0], &activations[0], (sizeof(double) * layer.size()), hipMemcpyDeviceToHost);
		hipFree(activations);
		hipFree(deviceNeurons);
		free(neuronBuffer);
		
		vector<double> result(&output[0], &output[layer.size()]);
		free(output);
		hipFree(connections);
		return result;
	} else return vector<double>();
}

vector<double> LSTMNetwork::train(vector<double> input, vector<double> target) {
	double *output = (double *)malloc(blocks.size() * blocks[0].nCells * sizeof(double)),
			*connections;
	hipMalloc((void **)&connections, sizeof(double) * input.size());
	hipMemcpy(&connections[0], &input[0], (sizeof(double) * input.size()), hipMemcpyHostToDevice);
	if (input.size() != inputSize) {
	    cout << "Target size mismatch" << endl;
		return vector<double>();
	}
    // start forward pass
    double *activations;
    hipMalloc((void **)&activations, (sizeof(double) * blocks.size() * blocks[0].nCells));
    MemoryBlock **deviceBlocks;
    for (int i = 0; i < blocks.size(); i++) {
        hipMemcpy(&(blocks[i].impulse[0]), &connections[0], (sizeof(double) * blocks[i].nConnections), hipMemcpyDeviceToHost);
    } hipMalloc((void **)&deviceBlocks, sizeof(MemoryBlock *) * blocks.size());
    for (int i = 0; i < blocks.size(); i++) {
        MemoryBlock *db = MemoryBlock::copyToGPU(&blocks[i]);
        hipMemcpy(&deviceBlocks[i], &db, sizeof(MemoryBlock *), hipMemcpyHostToDevice);
    } forwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, connections, activations, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
    hipDeviceSynchronize();
    hipFree(connections);
    hipMalloc((void **)&connections, (sizeof(double) * blocks.size() * blocks[0].nCells));
    hipMemcpy(&connections[0], &activations[0], (sizeof(double) * blocks.size() * blocks[0].nCells), hipMemcpyDeviceToDevice);
    hipFree(activations);
    free(output);

    cout << blocks.size() * blocks[0].nCells;
    for (int i = 0; i < blocks.size(); i++)

        cout << activations[i];

    output = (double *)malloc(sizeof(double) * layer.size());

    hipMalloc((void **)&activations, (sizeof(double) * layer.size()));

    Neuron **layerNeurons;
    for (int j = 0; j < layer.size(); j++) {
        hipMemcpy(&(layer[j].impulse[0]), &connections[0], (sizeof(double) * layer[j].connections), hipMemcpyDeviceToHost);
    }
    hipMalloc((void **)&layerNeurons, sizeof(Neuron *) * layer.size());
    for (int j = 0; j < layer.size(); j++) {
        Neuron *dn = Neuron::copyToGPU(&layer[j]);
        hipMemcpy(&layerNeurons[j], &dn, sizeof(Neuron *), hipMemcpyHostToDevice);
    }
    forwardPass<<<maxBlocks, maxThreads>>>(layerNeurons, connections, activations, layer.size(), ceil((double)layer.size() / (double)(maxBlocks * maxThreads)));
    hipDeviceSynchronize();
    hipFree(connections);
    hipMalloc((void **)&connections, (sizeof(double) * layer.size()));

    hipMemcpy(&connections[0], &activations[0], (sizeof(double) * layer.size()), hipMemcpyDeviceToDevice);
    hipFree(activations);
    hipFree(connections);



    // start backward pass
    double *weightedError;
    hipMalloc((void **)&weightedError, (sizeof(double) * layer.size()));
    for (int i = 0; i < layer.size(); i++) {
        double error = (output[i] - target[i]);
        output[i] = error;
        hipMemcpy(&weightedError[i], &error, sizeof(double), hipMemcpyHostToDevice);

    }
    double *errorSum;
    hipMalloc((void **)&errorSum, (sizeof(double) * layer[0].connections));
    hipMemset(&errorSum[0], 0, (sizeof(double) * layer[0].connections));

    // compute the gradient
    backwardPass<<<maxBlocks, maxThreads>>>(layerNeurons, weightedError, errorSum, learningRate, layer[0].connections, layer.size(), ceil((double)layer.size() / (double)(maxBlocks * maxThreads)));
    hipDeviceSynchronize();
    hipFree(weightedError);
    hipMalloc((void **)&weightedError, (sizeof(double) * layer[0].connections));
    hipMemcpy(&weightedError[0], &errorSum[0], (sizeof(double) * layer[0].connections), hipMemcpyDeviceToDevice);

    Neuron **neuronBuffer = (Neuron **)malloc(sizeof(Neuron) * layer.size());
    hipMemcpy(&neuronBuffer[0], &layerNeurons, (sizeof(Neuron *) * layer.size()), hipMemcpyDeviceToHost);
    for (int j = 0; j < layer.size(); j++) {
        layer[j] = *Neuron::copyFromGPU(neuronBuffer[j]);
    } free(neuronBuffer);
    hipFree(layerNeurons);
    hipFree(errorSum);


    double **errorChunks;
    hipMalloc((void **)&errorChunks, (sizeof(double *) * blocks.size()));
    hipMalloc((void **)&errorSum, (sizeof(double) * blocks[0].nConnections));
    hipMemset(&errorSum[0], 0.0, (sizeof(double) * blocks[0].nConnections));
    for (int i = 0; i < (blocks.size()); i++) {
        double *chunk;
        hipMalloc((void **)&chunk, (sizeof(double) * blocks[i].nCells));
        hipMemcpy(&chunk[0], &weightedError[(i * blocks[i].nCells)], (sizeof(double) * blocks[i].nCells), hipMemcpyDeviceToDevice);
        hipMemcpy(&errorChunks[i], &chunk, (sizeof(double *)), hipMemcpyHostToDevice);
    } backwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, errorChunks, errorSum, learningRate, blocks[0].nConnections, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
    hipDeviceSynchronize();

    MemoryBlock **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
    //cout << blocks.size() << " copy blocks " <<
    hipMemcpy(blockBuffer, deviceBlocks, (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);

    for (int i = 0; i < blocks.size(); i++) {
        MemoryBlock temp = *MemoryBlock::copyFromGPU(blockBuffer[i]);
        blocks[i] = temp;
    }

    hipFree(deviceBlocks);
    hipFree(weightedError);
    hipFree(errorChunks);
    hipFree(errorSum);

    vector<double> result(&output[0], &output[layer.size()]);
    free(output);
    return result;
}
