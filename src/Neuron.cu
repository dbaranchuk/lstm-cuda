#include "hip/hip_runtime.h"
#include "Neuron.cuh"

long long Neuron::n = 0;

Neuron::Neuron(int size) {
	activation = 0; activationPrime = 0;
	connections = size;
	default_random_engine g(time(0) + (n++));
	normal_distribution<double> d(0, 1);
	weightedError = (double *)malloc(sizeof(double) * size);
	weight = (double *)malloc(sizeof(double) * size);
	impulse = (double *)calloc(size, sizeof(double));
	for (int i = 0; i < size; i++) {
		weight[i] = (d(g));
	}
}

Neuron::~Neuron() {}

__device__ double Neuron::sigmoid(double input) {
	return 1 / (1 + exp(-input));
}

__device__ double Neuron::sigmoidPrime(double input) {
	return sigmoid(input) * (1 - sigmoid(input));
}

__device__ double Neuron::activate(double input) {
	return tanh(input);
}

__device__ double Neuron::activatePrime(double input) {
	return (1 - (tanh(input) * tanh(input)));
}

__device__ double Neuron::forward(double *input) {
	double sum = 0;
	// find the weighted sum of all input
	for (int i = 0; i < connections; i++) {
		sum += input[i] * weight[i];
	}
	activation = activate(sum);
	activationPrime = activatePrime(sum);
	return activation;
}

__device__ double *Neuron::backward(double errorPrime, double learningRate) {
	// update all weights
	for (int i = 0; i < connections; i++) {
		weightedError[i] = (errorPrime * weight[i] * activationPrime);
		weight[i] -= learningRate * errorPrime * impulse[i];
	}
	return weightedError;
}

Neuron *Neuron::copyToGPU(Neuron *data) {
	Neuron *neuron;
	hipMalloc((void **)&neuron, (sizeof(Neuron)));
	hipDeviceSynchronize();
	hipMemcpy(neuron, data, sizeof(Neuron), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	double *e;
	double *w;
	double *i;
	hipMalloc((void **)&e, (sizeof(double) * data->connections));
	hipMalloc((void **)&w, (sizeof(double) * data->connections));
	hipMalloc((void **)&i, (sizeof(double) * data->connections));
	hipDeviceSynchronize();

	hipMemcpy(e, data->weightedError, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipMemcpy(w, data->weight, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipMemcpy(i, data->impulse, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipMemcpy(&(neuron->weightedError), &e, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(neuron->weight), &w, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(neuron->impulse), &i, sizeof(double *), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	return neuron;
}

Neuron *Neuron::copyFromGPU(Neuron *data) {
	Neuron *neuron;
	neuron = (Neuron *)malloc((sizeof(Neuron)));
	hipDeviceSynchronize();
	hipMemcpy(neuron, data, sizeof(Neuron), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	double *e;
	double *w;
	double *i;
	e = (double *)malloc(sizeof(double) * neuron->connections);
	w = (double *)malloc(sizeof(double) * neuron->connections);
	i = (double *)malloc(sizeof(double) * neuron->connections);
	hipDeviceSynchronize();

	hipMemcpy(e, neuron->weightedError, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipMemcpy(w, neuron->weight, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipMemcpy(i, neuron->impulse, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	memcpy(&(neuron->weightedError), &e, sizeof(double *));
	memcpy(&(neuron->weight), &w, sizeof(double *));
	memcpy(&(neuron->impulse), &i, sizeof(double *));
	hipDeviceSynchronize();

	return neuron;
}

