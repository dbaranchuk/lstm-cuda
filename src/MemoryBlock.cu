#include "MemoryBlock.cuh"

long long int MemoryBlock::n = 0;

MemoryBlock::MemoryBlock(int cl, int cn) {
	nConnections = cn;
	nCells = cl;
	input = 0; inputPrime = 0;
	forget = 0; forgetPrime = 0;
	output = 0; outputPrime = 0;

	default_random_engine g(time(0) + (n++));
	normal_distribution<double> d(0, 1);

	bias = (double *)calloc(3, sizeof(double));
	cells = (MemoryCell **)malloc(sizeof(MemoryCell *) * nCells);
	inputFeedbackWeight = (double *)malloc(sizeof(double) * nCells);
	forgetFeedbackWeight = (double *)malloc(sizeof(double) * nCells);
	outputFeedbackWeight = (double *)malloc(sizeof(double) * nCells);

	for (int i = 0; i < nCells; i++) {
		cells[i] = (new MemoryCell(nConnections));
		inputFeedbackWeight[i] = (d(g));
		forgetFeedbackWeight[i] = (d(g));
		outputFeedbackWeight[i] = (d(g));
	}

	impulse = (double *)malloc(sizeof(double) * nConnections);
	inputDataWeight = (double *)malloc(sizeof(double) * nConnections);
	forgetDataWeight = (double *)malloc(sizeof(double) * nConnections);
	outputDataWeight = (double *)malloc(sizeof(double) * nConnections);

	for (int i = 0; i < nConnections; i++) {
		impulse[i] = (0);
		inputDataWeight[i] = (d(g));
		forgetDataWeight[i] = (d(g));
		outputDataWeight[i] = (d(g));
	}
}

MemoryBlock::~MemoryBlock() {
}


__device__ double MemoryBlock::inputGate(double data) {
	input = sigmoid(data);
	inputPrime = sigmoidPrime(data);
	return input;
}

__device__ double MemoryBlock::forgetGate(double data) {
	forget = sigmoid(data);
	forgetPrime = sigmoidPrime(data);
	return forget;
}

__device__ double MemoryBlock::outputGate(double data) {
	output = sigmoid(data);
	outputPrime = sigmoidPrime(data);
	return output;
}

__device__ double *MemoryBlock::forward(double *input) {
	double *cellSum = new double[nCells] {0};
	double inputSum = bias[0];
	double forgetSum = bias[1];
	double outputSum = bias[2];

	for (int i = 0; i < nCells; i++) {
		inputSum += (inputFeedbackWeight[i] * cells[i]->feedback);
		forgetSum += (forgetFeedbackWeight[i] * cells[i]->feedback);
		outputSum += (outputFeedbackWeight[i] * cells[i]->feedback);
	}

	// find the weighted sum of all input
	for (int i = 0; i < nConnections; i++) {
		for (unsigned int j = 0; j < nCells; j++) {
			cellSum[j] += input[i] * cells[j]->cellDataWeight[i];
		}
		inputSum += input[i] * inputDataWeight[i];
		forgetSum += input[i] * forgetDataWeight[i];
		outputSum += input[i] * outputDataWeight[i];
	}

	// compute input into memory
	double *output = new double[nCells];	// potential error
	for (int i = 0; i < nCells; i++) {
		cells[i]->previousState = cells[i]->state;
		cells[i]->state *= forgetGate(forgetSum);
		cells[i]->state += cells[i]->activateIn(cellSum[i]) * inputGate(inputSum);

		// compute output of memory cell
		cells[i]->previousFeedback = cells[i]->feedback;
		cells[i]->feedback = cells[i]->activateOut(cells[i]->state) * outputGate(outputSum);
		output[i] = (cells[i]->feedback);
	}

	return output;
}

// errorprime must be a vector with length of number of cells
__device__ double *MemoryBlock::backward(double *errorPrime, double learningRate) {
	double *eta = new double[nCells],
			*inputDataPartialSum = new double[nConnections] {0},
			*forgetDataPartialSum = new double[nConnections] {0};
	double blockSum = 0,
			inputFeedbackPartialSum = 0,
			forgetFeedbackPartialSum = 0;

	for (int i = 0; i < nCells; i++) {
		blockSum += cells[i]->activationOut * errorPrime[i];
		eta[i] = (output * cells[i]->activationOutPrime * errorPrime[i]);
		outputFeedbackWeight[i] -= learningRate * blockSum * outputPrime * cells[i]->feedback;
	}

	for (int i = 0; i < nConnections; i++) {
		outputDataWeight[i] -= learningRate * blockSum * outputPrime * impulse[i];	// invalid read of size 8
	}

	// calculate the updates, and update the cell weights
	for (int i = 0; i < nCells; i++) {
		for (int j = 0; j < nConnections; j++) {
			cells[i]->cellDataPartial[j] = cells[i]->cellDataPartial[j] * forget + cells[i]->activationInPrime * input * impulse[j];
			cells[i]->cellDataWeight[j] -= learningRate * eta[i] * cells[i]->cellDataPartial[j];
			cells[i]->forgetDataPartial[j] = cells[i]->forgetDataPartial[j] * forget + cells[i]->previousState * forgetPrime * impulse[j];	// invalid read of size 8
			cells[i]->inputDataPartial[j] = cells[i]->inputDataPartial[j] * forget + cells[i]->activationIn * inputPrime * impulse[j];	// invalid read of size 8
			forgetDataPartialSum[j] += cells[i]->forgetDataPartial[j] * eta[i];
			inputDataPartialSum[j] += cells[i]->inputDataPartial[j] * eta[i];
		}

		cells[i]->cellFeedbackPartial = cells[i]->cellFeedbackPartial * forget + cells[i]->activationInPrime * input * cells[i]->previousFeedback;
		cells[i]->cellFeedbackWeight -= learningRate * eta[i] * cells[i]->cellFeedbackPartial;

		cells[i]->forgetFeedbackPartial = cells[i]->forgetFeedbackPartial * forget + cells[i]->previousState * forgetPrime * cells[i]->previousFeedback;
		forgetFeedbackPartialSum += eta[i] *cells[i]->forgetFeedbackPartial;

		cells[i]->inputFeedbackPartial = cells[i]->inputFeedbackPartial * forget + cells[i]->activationIn * inputPrime * cells[i]->previousFeedback;
		inputFeedbackPartialSum += eta[i] *cells[i]->inputFeedbackPartial;
	}

	// update the input, output, and forget weights
	for (int i = 0; i < nCells; i++) {
		for (int j = 0; j < nConnections; j++) {
			forgetDataWeight[j] -= learningRate * forgetDataPartialSum[j];	// invalid read of size 8
			inputDataWeight[j] -= learningRate * inputDataPartialSum[j];	// invalid read of size 8
		}
		inputFeedbackWeight[i] -= learningRate * inputFeedbackPartialSum;
		forgetFeedbackWeight[i] -= learningRate * forgetFeedbackPartialSum;
	}

	double *temp = new double[nConnections];	// potential error
	for (int i = 0; i < nConnections; i++) {
		temp[i] = (0.0);
	}


	return temp;
}

MemoryBlock *MemoryBlock::copyToGPU(MemoryBlock *memory) {
	MemoryBlock *memoryBlock;
	hipMalloc((void **)&memoryBlock, (sizeof(MemoryBlock)));
	hipDeviceSynchronize();
	hipMemcpy(memoryBlock, memory, sizeof(MemoryBlock), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	MemoryCell **memoryCells;
	hipMalloc((void **)&memoryCells, ((sizeof(MemoryCell *) * memory->nCells)));
	for (int i = 0; i < memory->nCells; i++) {
		MemoryCell *buffer = MemoryCell::copyToGPU(memory->cells[i]);
		hipMemcpy(&memoryCells[i], &buffer, sizeof(MemoryCell *), hipMemcpyHostToDevice);
	} hipMemcpy(&(memoryBlock->cells), &memoryCells, sizeof(MemoryCell **), hipMemcpyHostToDevice);


	double *ifw, *ffw, *ofw, *b;
	hipMalloc((void **)&ifw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&ffw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&ofw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&b, (sizeof(double) * 3));

	double *idw, *fdw, *odw, *i;
	hipMalloc((void **)&idw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&fdw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&odw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&i, (sizeof(double) * memory->nConnections));
	hipDeviceSynchronize();

	hipMemcpy(ifw, memory->inputFeedbackWeight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(ffw, memory->forgetFeedbackWeight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(ofw, memory->outputFeedbackWeight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(b, memory->bias, (sizeof(double) * 3), hipMemcpyHostToDevice);
	hipMemcpy(idw, memory->inputDataWeight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(fdw, memory->forgetDataWeight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(odw, memory->outputDataWeight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(i, memory->impulse, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipMemcpy(&(memoryBlock->inputFeedbackWeight), &(ifw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->forgetFeedbackWeight), &(ffw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->outputFeedbackWeight), &(ofw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->bias), &(b), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->inputDataWeight), &(idw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->forgetDataWeight), &(fdw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->outputDataWeight), &(odw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->impulse), &(i), (sizeof(double *)), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	return memoryBlock;
}

MemoryBlock *MemoryBlock::copyFromGPU(MemoryBlock *memory) {

	MemoryBlock *memoryBlock;
	memoryBlock = (MemoryBlock *)malloc((sizeof(MemoryBlock)));
	hipDeviceSynchronize();
	hipMemcpy(memoryBlock, memory, sizeof(MemoryBlock), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	MemoryCell **memoryCells;
	memoryCells = (MemoryCell **)malloc((sizeof(MemoryCell *) * memoryBlock->nCells));
	hipMemcpy(memoryCells, memoryBlock->cells, (sizeof(MemoryCell *) * memoryBlock->nCells), hipMemcpyDeviceToHost);

	for (int i = 0; i < memoryBlock->nCells; i++) {
		MemoryCell *buffer = MemoryCell::copyFromGPU(memoryCells[i]);
		memoryCells[i] = buffer;
	} memcpy(&(memoryBlock->cells), &memoryCells, sizeof(MemoryCell *));


	double *ifw, *ffw, *ofw, *b;
	ifw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ifw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ffw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ofw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	b = (double *)malloc((sizeof(double) * 3));

	double *idw, *fdw, *odw, *i;
	idw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	fdw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	odw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	i = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	hipDeviceSynchronize();

	hipMemcpy(ifw, memoryBlock->inputFeedbackWeight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(ffw, memoryBlock->forgetFeedbackWeight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(ofw, memoryBlock->outputFeedbackWeight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(b, memoryBlock->bias, (sizeof(double) * 3), hipMemcpyDeviceToHost);
	hipMemcpy(idw, memoryBlock->inputDataWeight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(fdw, memoryBlock->forgetDataWeight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(odw, memoryBlock->outputDataWeight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(i, memoryBlock->impulse, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	memcpy(&(memoryBlock->inputFeedbackWeight), &ifw, (sizeof(double *)));
	memcpy(&(memoryBlock->forgetFeedbackWeight), &ffw, (sizeof(double *)));
	memcpy(&(memoryBlock->outputFeedbackWeight), &ofw, (sizeof(double *)));
	memcpy(&(memoryBlock->bias), &b, (sizeof(double *)));
	memcpy(&(memoryBlock->inputDataWeight), &idw, (sizeof(double *)));
	memcpy(&(memoryBlock->forgetDataWeight), &fdw, (sizeof(double *)));
	memcpy(&(memoryBlock->outputDataWeight), &odw, (sizeof(double *)));
	memcpy(&(memoryBlock->impulse), &i, (sizeof(double *)));

	return memoryBlock;
}

