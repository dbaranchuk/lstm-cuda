#include "hip/hip_runtime.h"
#include "LSTMCell.cuh"

long long int LSTMCell::n = 0;

LSTMCell::LSTMCell(int output_size, int input_size) {
	nConnections = input_size;
	nCells = output_size;
	input = 0; inputPrime = 0;
	forget = 0; forgetPrime = 0;
	output = 0; outputPrime = 0;

	default_random_engine g(time(0) + (n++));
	normal_distribution<double> d(0, 1);

	bias = (double *)calloc(3, sizeof(double));
	cells = (MemoryCell **)malloc(sizeof(MemoryCell *) * output_size);
	input_hidden_weight = (double *)malloc(sizeof(double) * output_size);
	forget_hidden_weight = (double *)malloc(sizeof(double) * output_size);
	output_hidden_weight = (double *)malloc(sizeof(double) * output_size);

	for (int i = 0; i < nCells; i++) {
		cells[i] = (new MemoryCell(output_size));
		input_hidden_weight[i] = d(g);
		forget_hidden_weight[i] = d(g);
		output_hidden_weight[i] = d(g);
	}

	//impulse = (double *)malloc(sizeof(double) * nConnections);
	input_data_weight = (double *)malloc(sizeof(double) * nConnections);
	forget_data_weight = (double *)malloc(sizeof(double) * nConnections);
	output_data_weight = (double *)malloc(sizeof(double) * nConnections);

	for (int i = 0; i < nConnections; i++) {
		//impulse[i] = 0;
		input_data_weight[i] = d(g);
		forget_data_weight[i] = d(g);
		output_data_weight[i] = d(g);
	}
}

LSTMCell::~LSTMCell() {
}


__device__ double LSTMCell::sigmoid(double input) {
	return 1 / (1 + exp(-input));
}

__device__ double LSTMCell::sigmoidPrime(double input) {
	return sigmoid(input) * (1 - sigmoid(input));
}

__device__ double LSTMCell::inputGate(double data) {
	input = sigmoid(data);
	inputPrime = sigmoidPrime(data);
	return input;
}

__device__ double LSTMCell::forgetGate(double data) {
	forget = sigmoid(data);
	forgetPrime = sigmoidPrime(data);
	return forget;
}

__device__ double LSTMCell::outputGate(double data) {
	output = sigmoid(data);
	outputPrime = sigmoidPrime(data);
	return output;
}


LSTMCell *LSTMCell::copyToGPU(LSTMCell *memory) {
	LSTMCell *memoryBlock;
	hipMalloc((void **)&memoryBlock, (sizeof(LSTMCell)));
	hipDeviceSynchronize();
	hipMemcpy(memoryBlock, memory, sizeof(LSTMCell), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	MemoryCell **memoryCells;
	hipMalloc((void **)&memoryCells, ((sizeof(MemoryCell *) * memory->nCells)));
	for (int i = 0; i < memory->nCells; i++) {
		MemoryCell *buffer = MemoryCell::copyToGPU(memory->cells[i]);
		hipMemcpy(&memoryCells[i], &buffer, sizeof(MemoryCell *), hipMemcpyHostToDevice);
	} hipMemcpy(&(memoryBlock->cells), &memoryCells, sizeof(MemoryCell **), hipMemcpyHostToDevice);


	double *ifw, *ffw, *ofw, *b;
	hipMalloc((void **)&ifw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&ffw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&ofw, (sizeof(double) * memory->nCells));
	hipMalloc((void **)&b, (sizeof(double) * 3));

	double *idw, *fdw, *odw, *i;
	hipMalloc((void **)&idw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&fdw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&odw, (sizeof(double) * memory->nConnections));
	hipMalloc((void **)&i, (sizeof(double) * memory->nConnections));
	hipDeviceSynchronize();

	hipMemcpy(ifw, memory->input_hidden_weight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(ffw, memory->forget_hidden_weight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(ofw, memory->output_hidden_weight, (sizeof(double) * memory->nCells), hipMemcpyHostToDevice);
	hipMemcpy(b, memory->bias, (sizeof(double) * 3), hipMemcpyHostToDevice);
	hipMemcpy(idw, memory->input_data_weight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(fdw, memory->forget_data_weight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipMemcpy(odw, memory->output_data_weight, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	//hipMemcpy(i, memory->impulse, (sizeof(double) * memory->nConnections), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipMemcpy(&(memoryBlock->input_hidden_weight), &(ifw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->forget_hidden_weight), &(ffw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->output_hidden_weight), &(ofw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->bias), &(b), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->input_data_weight), &(idw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->forget_data_weight), &(fdw), (sizeof(double *)), hipMemcpyHostToDevice);
	hipMemcpy(&(memoryBlock->output_data_weight), &(odw), (sizeof(double *)), hipMemcpyHostToDevice);
	//hipMemcpy(&(memoryBlock->impulse), &(i), (sizeof(double *)), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	return memoryBlock;
}

LSTMCell *LSTMCell::copyFromGPU(LSTMCell *memory) {

	LSTMCell *memoryBlock;
	memoryBlock = (LSTMCell *)malloc((sizeof(LSTMCell)));
	hipDeviceSynchronize();
	hipMemcpy(memoryBlock, memory, sizeof(LSTMCell), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	MemoryCell **memoryCells;
	memoryCells = (MemoryCell **)malloc((sizeof(MemoryCell *) * memoryBlock->nCells));
	hipMemcpy(memoryCells, memoryBlock->cells, (sizeof(MemoryCell *) * memoryBlock->nCells), hipMemcpyDeviceToHost);

	for (int i = 0; i < memoryBlock->nCells; i++) {
		MemoryCell *buffer = MemoryCell::copyFromGPU(memoryCells[i]);
		memoryCells[i] = buffer;
	} memcpy(&(memoryBlock->cells), &memoryCells, sizeof(MemoryCell *));


	double *ifw, *ffw, *ofw, *b;
	ifw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ifw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ffw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	ofw = (double *)malloc((sizeof(double) * memoryBlock->nCells));
	b = (double *)malloc((sizeof(double) * 3));

	double *idw, *fdw, *odw, *i;
	idw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	fdw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	odw = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	i = (double *)malloc((sizeof(double) * memoryBlock->nConnections));
	hipDeviceSynchronize();

	hipMemcpy(ifw, memoryBlock->input_hidden_weight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(ffw, memoryBlock->forget_hidden_weight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(ofw, memoryBlock->output_hidden_weight, (sizeof(double) * memoryBlock->nCells), hipMemcpyDeviceToHost);
	hipMemcpy(b, memoryBlock->bias, (sizeof(double) * 3), hipMemcpyDeviceToHost);
	hipMemcpy(idw, memoryBlock->input_data_weight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(fdw, memoryBlock->forget_data_weight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipMemcpy(odw, memoryBlock->output_data_weight, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	//hipMemcpy(i, memoryBlock->impulse, (sizeof(double) * memoryBlock->nConnections), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	memcpy(&(memoryBlock->input_hidden_weight), &ifw, (sizeof(double *)));
	memcpy(&(memoryBlock->forget_hidden_weight), &ffw, (sizeof(double *)));
	memcpy(&(memoryBlock->output_hidden_weight), &ofw, (sizeof(double *)));
	memcpy(&(memoryBlock->bias), &b, (sizeof(double *)));
	memcpy(&(memoryBlock->input_data_weight), &idw, (sizeof(double *)));
	memcpy(&(memoryBlock->forget_data_weight), &fdw, (sizeof(double *)));
	memcpy(&(memoryBlock->output_data_weight), &odw, (sizeof(double *)));
	//memcpy(&(memoryBlock->impulse), &i, (sizeof(double *)));

	return memoryBlock;
}

