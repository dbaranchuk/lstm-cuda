#include "hip/hip_runtime.h"
#include "TextClassifier.cuh"

__global__ void forwardPass(Neuron **neurons, double *connections, double *activations, int size) {
	//int maxId = gridDim.x * blockDim.x;
	//int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
	//if (idx < size) {
	for (int i = 0; i < size; i++)
		activations[i] = neurons[i]->forward(connections);
	//}
}

//__global__ void backwardPass(Neuron **neurons, double *weightedError, double *errorSum,
//							 double learningRate, int connections, int size) {
//	int maxId = gridDim.x * blockDim.x;
//	int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
//	if (idx < size) {
//		double *contribution = neurons[idx]->backward(weightedError[idx], learningRate);
//		for (int j = 0; j < connections; j++) {
//			errorSum[j] += contribution[j];
//		}
//	}
//}

//__global__ void forwardPassLSTM(MemoryBlock **blocks, double *connections, double *activations, int size, int cycles) {
//	int maxId = gridDim.x * blockDim.x;
//	for (int i = 0; i < (cycles); i++) {
//		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
//		if (idx < size) {
//			double *blockActivation = blocks[idx]->forward(connections);
//			for (int j = 0; j < blocks[i]->nCells; j++) activations[idx * blocks[i]->nCells + j] = blockActivation[j];
//		}
//	}
//}

__global__ void forwardPassLSTM(MemoryBlock *block, double **connections, double *activations, int cycles) {
    double *local_activations;
    for (int i = 0; i < cycles; i++) {
		local_activations = block->forward(connections[i]);
	}
    for (int i = 0; i < block->nCells; i++)
        activations[i] = local_activations[i];
}

//__global__ void backwardPassLSTM(MemoryBlock **blocks, double **weightedError, double *errorSum, double learningRate, int connections, int size, int cycles) {
//	int maxId = gridDim.x * blockDim.x;
//	for (int i = 0; i < (cycles); i++) {
//		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
//		if (idx < size) {
//			double *contribution = blocks[idx]->backward(weightedError[idx], learningRate);
//			for (int j = 0; j < connections; j++) {
//				errorSum[j] += contribution[j];
//			}
//		}
//	}
//}

TextClassifier::TextClassifier(int is, int c, double lr, int num_classes) {
	inputSize = is;
	learningRate = lr;
	block = new MemoryBlock(c, is);

	layer = vector<Neuron>;
	for (int i = 0; i < num_classes; i++)
		layer.push_back(Neuron(c));
}

TextClassifier::~TextClassifier() {}

//vector<double> TextClassifier::classify(vector<double> input) {
//	double *connections;
//	hipMalloc((void **)&connections, sizeof(double) * input.size());
//	hipMemcpy(&connections[0], &input[0], (sizeof(double) * input.size()), hipMemcpyHostToDevice);
//	if (input.size() == inputSize) {
//		// calculate activations from bottom up
//		double *activations;
//		hipMalloc((void **)&activations, (sizeof(double) * blocks.size() * blocks[0].nCells));
//
//		MemoryBlock **deviceBlocks, **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
//		for (int i = 0; i < blocks.size(); i++) {
//			hipMemcpy(&(blocks[i].impulse[0]), &connections[0], (sizeof(double) * blocks[i].nConnections), hipMemcpyDeviceToHost);
//		}
//		hipMalloc((void **)&deviceBlocks, sizeof(MemoryBlock *) * blocks.size());
//		for (int i = 0; i < blocks.size(); i++) {
//			MemoryBlock *db = MemoryBlock::copyToGPU(&blocks[i]);
//			hipMemcpy(&deviceBlocks[i], &db, sizeof(MemoryBlock *), hipMemcpyHostToDevice);
//		} forwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, connections, activations, blocks.size(),
//		                                             ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
//		hipDeviceSynchronize();
//
//		hipMemcpy(&blockBuffer[0], &deviceBlocks[0], (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);
//		for (int i = 0; i < blocks.size(); i++) {
//			blocks[i] = *MemoryBlock::copyFromGPU(blockBuffer[i]);
//		} free(blockBuffer);
//		hipFree(deviceBlocks);
//
//		hipFree(connections);
//		hipMalloc((void **)&connections, (sizeof(double) * blocks.size() * blocks[0].nCells));
//		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * blocks.size() * blocks[0].nCells), hipMemcpyDeviceToDevice);
//		hipFree(activations);
//
//        // logits_layer
//		hipMalloc((void **)&activations, (sizeof(double) * logits_layer.size()));
//
//		Neuron **deviceNeurons, **neuronBuffer = (Neuron **)malloc(sizeof(Neuron *) * logits_layer.size());
//		for (int j = 0; j < logits_layer.size(); j++) {
//			hipMemcpy(&(layer[j].impulse[0]), &connections[0], (sizeof(double) * logits_layer[j].connections), hipMemcpyDeviceToHost);
//		}
//		hipMalloc((void **)&deviceNeurons, sizeof(Neuron *) * logits_layer.size());
//		for (int j = 0; j < logits_layer.size(); j++) {
//			Neuron *dn = Neuron::copyToGPU(&layer[j]);
//			hipMemcpy(&deviceNeurons[j], &dn, sizeof(Neuron *), hipMemcpyHostToDevice);
//		} forwardPass<<<maxBlocks, maxThreads>>>(deviceNeurons, connections, activations, logits_layer.size(), ceil((double)layer.size() / (double)(maxBlocks * maxThreads)));
//		hipDeviceSynchronize();
//
//		hipFree(connections);
//		hipMalloc((void **)&connections, (sizeof(double) * logits_layer.size()));
//		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * logits_layer.size()), hipMemcpyDeviceToDevice);
//		hipMemcpy(&neuronBuffer[0], &deviceNeurons[0], (sizeof(Neuron *) * logits_layer.size()), hipMemcpyDeviceToHost);
//		for (int j = 0; j < logits_layer.size(); j++) {
//			layer[j] = *Neuron::copyFromGPU(neuronBuffer[j]);
//		}
//		double *output = (double *)malloc(sizeof(double) * logits_layer.size());
//		hipMemcpy(&output[0], &activations[0], (sizeof(double) * logits_layer.size()), hipMemcpyDeviceToHost);
//		hipFree(activations);
//		hipFree(deviceNeurons);
//		free(neuronBuffer);
//
//		vector<double> result(&output[0], &output[layer.size()]);
//		free(output);
//		hipFree(connections);
//		return result;
//	} else return vector<double>();
//}

double TextClassifier::train(vector<vector<double>> &inputs, vector<double> &target) {
	if (inputs[0].size() != inputSize) {
	    cout << "Target size mismatch" << endl;
		return 0.0;
	}
    // Load input data to GPU
    double **connections;
	double *lstm_activations;
	hipMalloc((void **)&connections, sizeof(double *) * inputs.size());
    hipMalloc((void **)&lstm_activations, sizeof(double) * block.nCells);
    for (int i = 0; i < inputs.size(); i++) {
        hipMalloc((void **) &connections[i], sizeof(double) * inputs[0].size());
        hipMemcpy(connections[i].data(), inputs[i].data(),
                   sizeof(double) * inputs[0].size(), hipMemcpyHostToDevice);
    }
	cout << inputs[0].size() << " " << block.nConnections;
	// TODO
	for (int i = 0; i < inputs.size(); i++) {
        hipMemcpy(block.impulses[i].data(), connections[i].data(),
                   (sizeof(double) * block.nConnections), hipMemcpyDeviceToHost);
    }
    MemoryBlock *device_block = MemoryBlock::copyToGPU(block);
    forwardPassLSTM<<<maxBlocks, maxThreads>>>(device_block, connections, lstm_activations, inputs.size());
    hipDeviceSynchronize();
	hipFree(connections);

    // lstm_activations become new connections for logit logits_layer

	double *logits_activations;
	hipMalloc((void **)&logits_activations, sizeof(double) * logits_layer.size());

	// Logits

	// Put lstm activation to impulse for backprop
    Neuron **layerNeurons;
    for (int j = 0; j < logits_layer.size(); j++) {
        hipMemcpy(logits_layer[j].impulse.data(), lstm_activations.data(),
        		sizeof(double) * logits_layer[j].connections, hipMemcpyDeviceToHost);
    }
	// Copy linear logits_layer to device
    hipMalloc((void **)&layerNeurons, sizeof(Neuron *) * logits_layer.size());
    for (int j = 0; j < logits_layer.size(); j++) {
        Neuron *device_neuron = Neuron::copyToGPU(&logits_layer[j]);
        hipMemcpy(&layerNeurons[j], &device_neuron, sizeof(Neuron *), hipMemcpyHostToDevice);
    }
    
    // Logits forward
    forwardPass<<<maxBlocks, maxThreads>>>(layerNeurons, lstm_activations, 
    		                               logits_activations, logits_layer.size());
    hipDeviceSynchronize();
    hipFree(lstm_activations);
    
    double *output = (double *)malloc(sizeof(double) * logits_layer.size());
    hipMemcpy(output.data(), logits_activations.data(), 
    		   sizeof(double) * logits_layer.size(), hipMemcpyDeviceToHost);

    hipFree(logits_activations);

    cout << logits_layer.size() << "\n";
	double loss = 0.0;
    for (int i = 0; i < logits_layer.size(); i++)
        loss += output[i];

    return loss
    ///////////////////////////////////////////////////////////////

    // start backward pass
//    double *weightedError;
//    hipMalloc((void **)&weightedError, (sizeof(double) * logits_layer.size()));
//    for (int i = 0; i < logits_layer.size(); i++) {
//        double error = (output[i] - target[i]);
//        output[i] = error;
//        hipMemcpy(&weightedError[i], &error, sizeof(double), hipMemcpyHostToDevice);
//
//    }
//    double *errorSum;
//    hipMalloc((void **)&errorSum, (sizeof(double) * logits_layer[0].connections));
//    hipMemset(&errorSum[0], 0, (sizeof(double) * logits_layer[0].connections));
//
//    // compute the gradient
//    backwardPass<<<maxBlocks, maxThreads>>>(layerNeurons, weightedError, errorSum, learningRate, logits_layer[0].connections, logits_layer.size(), ceil((double)layer.size() / (double)(maxBlocks * maxThreads)));
//    hipDeviceSynchronize();
//    hipFree(weightedError);
//    hipMalloc((void **)&weightedError, (sizeof(double) * logits_layer[0].connections));
//    hipMemcpy(&weightedError[0], &errorSum[0], (sizeof(double) * logits_layer[0].connections), hipMemcpyDeviceToDevice);
//
//    Neuron **neuronBuffer = (Neuron **)malloc(sizeof(Neuron) * logits_layer.size());
//    hipMemcpy(&neuronBuffer[0], &layerNeurons, (sizeof(Neuron *) * logits_layer.size()), hipMemcpyDeviceToHost);
//    for (int j = 0; j < logits_layer.size(); j++) {
//        logits_layer[j] = *Neuron::copyFromGPU(neuronBuffer[j]);
//    } free(neuronBuffer);
//    hipFree(layerNeurons);
//    hipFree(errorSum);
//
//
//    double **errorChunks;
//    hipMalloc((void **)&errorChunks, (sizeof(double *) * blocks.size()));
//    hipMalloc((void **)&errorSum, (sizeof(double) * blocks[0].nConnections));
//    hipMemset(&errorSum[0], 0.0, (sizeof(double) * blocks[0].nConnections));
//    for (int i = 0; i < (blocks.size()); i++) {
//        double *chunk;
//        hipMalloc((void **)&chunk, (sizeof(double) * blocks[i].nCells));
//        hipMemcpy(&chunk[0], &weightedError[(i * blocks[i].nCells)], (sizeof(double) * blocks[i].nCells), hipMemcpyDeviceToDevice);
//        hipMemcpy(&errorChunks[i], &chunk, (sizeof(double *)), hipMemcpyHostToDevice);
//    } backwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, errorChunks, errorSum, learningRate, blocks[0].nConnections, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
//    hipDeviceSynchronize();
//
//    MemoryBlock **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
//    //cout << blocks.size() << " copy blocks " <<
//    hipMemcpy(blockBuffer, deviceBlocks, (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);
//
//    for (int i = 0; i < blocks.size(); i++) {
//        MemoryBlock temp = *MemoryBlock::copyFromGPU(blockBuffer[i]);
//        blocks[i] = temp;
//    }
//
//    hipFree(deviceBlocks);
//    hipFree(weightedError);
//    hipFree(errorChunks);
//    hipFree(errorSum);

//    vector<double> result(&output[0], &output[layer.size()]);
//    free(output);
//    return result;
}
